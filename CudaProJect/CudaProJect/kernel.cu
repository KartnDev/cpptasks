#include "hip/hip_runtime.h"
// Using CUDA device to calculate pi
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#define DP
#define TRIALS_PER_THREAD 4096
#define NUM_BLOCK  256  // Number of thread blocks
#define NUM_THREAD  256  // Number of threads per block
// #define NBIN TRIALS_PER_THREAD*NUM_THREAD*NUM_BLOCK  // Number of bins 4096*256*256
// 
// // 2^28
#define NBIN 268435456  // Number of bins 4096*256*256

// // 2^26
// #define NBIN 67108864  // Number of bins 1024*256*256

// 2^24
// #define NBIN 16777216  // Number of bins 256*256*256

//Help code for switching between Single Precision and Double Precision
#ifdef DP
typedef double Real;
#define PI  3.14159265358979323846  // known value of pi
#else
typedef float Real;
#define PI 3.1415926535  // known value of pi
#endif

int tid;
Real pi = 0;

// Kernel that executes on the CUDA device
__global__ void cal_pi(Real *sum, int nbin, Real step, int nthreads, int nblocks) {
	int i;
	Real x;
	int idx = blockIdx.x*blockDim.x + threadIdx.x;  // Sequential thread index across the blocks
	for (i = idx; i< nbin; i += nthreads*nblocks) {
		x = (i + 0.5)*step;
		sum[idx] += 4.0 / (1.0 + x*x);
	}
}

// Main routine that executes on the host
int main(void) {

	clock_t start, end;

	dim3 dimGrid(NUM_BLOCK, 1, 1);  // Grid dimensions
	dim3 dimBlock(NUM_THREAD, 1, 1);  // Block dimensions
	Real *sumHost, *sumDev;  // Pointer to host & device arrays

	printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d\n", TRIALS_PER_THREAD, NUM_BLOCK, NUM_THREAD);

	Real step = 1.0 / NBIN;  // Step size
	size_t size = NUM_BLOCK*NUM_THREAD * sizeof(Real);  //Array memory size
	sumHost = (Real *)malloc(size);  //  Allocate array on host

	start = clock();

	hipMalloc((void **)&sumDev, size);  // Allocate array on device
										 // Initialize array in device to 0
	hipMemset(sumDev, 0, size);
	// Do calculation on device
	cal_pi << <dimGrid, dimBlock >> > (sumDev, NBIN, step, NUM_THREAD, NUM_BLOCK); // call CUDA kernel
																				   // Retrieve result from device and store it in host array
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
	for (tid = 0; tid<NUM_THREAD*NUM_BLOCK; tid++)
		pi += sumHost[tid];
	pi *= step;

	end = clock();
	// Print results

	printf("GPU PI calculated in : %f s.\n", (end - start) / (float)CLOCKS_PER_SEC);

#ifdef DP
	printf("GPU estimated PI = %20.18f [error of %20.18f]\n", pi, pi - PI);
#else
	printf("GPU estimated PI = %f [error of %f]\n", pi, pi - PI);
#endif

	// Cleanup
	free(sumHost);
	hipFree(sumDev);
	getchar();
	return 0;
}